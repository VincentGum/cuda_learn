#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void reduce_baseline(const int* input, int* output, size_t n) {
  int sum = 0;
  for(int i = 0; i < n; i++) {
    sum += input[i];
  }
  *output = sum;
}

// void ReduceBySerial(const float* input, float* output, size_t n) {
//   SerialKernel<<<1, 1>>>(intput, output, n);
// }


bool checkResult(int *out, int groundTruth, int n){
    if (*out != groundTruth){
        return false;
    }
    return true;
}

int main() {
    float milliseconds = 0;
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    const int blockSize = 1;
    int gridSize = 1;
    
    // 分配内存，此处可以优化为UnifiedMemory
    int *a = (int *)malloc(N * sizeof(int));
    int *d_a;
    hipMalloc((void **)&d_a, N*sizeof(int));

    int *out = (int *)malloc(gridSize * sizeof(int));
    int *d_out;
    hipMalloc((void **)&d_out, gridSize * sizeof(int));

    // 初始化
    for(int i = 0; i < N; i++){
        a[i] = 1;
    }


    // 预期
    int groundTruth = N * 1;
    
    //拷贝数据到GPU
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    
    dim3 Grid(gridSize);
    dim3 Block(blockSize);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    reduce_baseline<<<gridSize, blockSize>>>(d_a, d_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, gridSize * sizeof(int), hipMemcpyDeviceToHost);
    printf("allocated %d blocks, data counts are %d\n", gridSize, N);

    bool is_right = checkResult(out, groundTruth, gridSize);
    if(is_right) {
        printf("RIGHT!\n");
    } else {
        printf("WRONG!\n");
        for(int i = 0; i < gridSize; i++){
            printf("res per block: %lf", out[i]); 
        }
    }

    return 0;
}